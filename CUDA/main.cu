#include "blowfish.h"
#include "implement.h"
#include "const.h"
#include <stdio.h>
#include <string.h>
#include <stdint.h>

int main(int argc, char *argv[])
{
	//splashscreen();
	version();
	// Misc variables
	int status = 0;
	uint64_t hash_original, hash_encrypted, hash_decrypted;
	float cudaRuntime, rate;
	// File variables
	size_t filesize;
	const char *filepath = "../testfile";
	uint32_t *file = (uint32_t*) readfile(&filesize, filepath);
	size_t numblocks = filesize/sizeof(uint32_t);
	printf("File size = %zu bytes, numblocks = %zu\n", filesize, numblocks/2);
	// Encryption key
	const char *key = "TESTKEY";
	printf("Key = %s, length = %zu\n", key, strlen(key));
	// Create Blowfish context
	blowfish_context_t *context = (blowfish_context_t*) malloc(sizeof(blowfish_context_t));
	if(!context) 
	{
		printf("Could not allocate enough memory!\n");
		return -1;
	}

	// Initialize key schedule
	status = blowfish_init(context, key, strlen(key));
	if (status)
	{
		printf("Error initiating key\n");
		return -1;
	} else printf("Key schedule complete!\n");

	// Hash original file
	hash_original = hash(file, numblocks);
	printf("Original hash = %llx\n", (unsigned long long)hash_original);

	// CUDA Starts
	printf("CUDA Starts!\n");

	int i;
	const int numStreams = NUM_STREAMS;
	const int streamSize = filesize/numStreams;
	const int numBlocksPerStream = numblocks/numStreams;

	hipStream_t stream[numStreams];

	for (i = 0; i < numStreams; ++i)
		hipStreamCreate(&stream[i]);

	uint32_t *filegpu;
	hipMalloc(&filegpu, filesize);
	// hipMemcpy(filegpu, file, filesize, hipMemcpyHostToDevice);

	blowfish_context_t *ctxgpu;
	hipMalloc(&ctxgpu, sizeof(blowfish_context_t));
	hipMemcpy(ctxgpu, context, sizeof(blowfish_context_t), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int blocksize = 1024;
	int gridsize = (numblocks/(blocksize<<1)+1)/numStreams;

	//__________ENCRYPTION__________
	printf("Encryption starts...\n");

	hipEventRecord(start);

	for (i = 0; i < numStreams; ++i)
	{
		int offset = i * numBlocksPerStream;
		hipMemcpyAsync(&filegpu[offset], &file[offset], streamSize, hipMemcpyHostToDevice, stream[i]);
		cudaBlowfishEncryptPtr<<<gridsize, blocksize, 0, stream[i]>>>(ctxgpu, &filegpu[offset], numBlocksPerStream);
		hipMemcpyAsync(&file[offset], &filegpu[offset], streamSize, hipMemcpyDeviceToHost, stream[i]);
	}

	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	hash_encrypted = hash(file, numblocks);

	hipEventElapsedTime(&cudaRuntime, start, stop);
	rate = (filesize*1e3)/(cudaRuntime);

	printf("Encryption done!\n");
	printf("Time taken: %lf milliseconds\n", cudaRuntime);
	printf("Average speed: %lf MB/s\n", rate/MEGABYTE);
	printf("Encrypted hash = %llx\n", (unsigned long long)hash_encrypted);

	//__________DECRYPTION__________
	printf("Decryption starts...\n");

	hipEventRecord(start);
	for (i = 0; i < numStreams; ++i)
	{
		int offset = i * numBlocksPerStream;
		hipMemcpyAsync(&filegpu[offset], &file[offset], streamSize, hipMemcpyHostToDevice, stream[i]);
		cudaBlowfishDecryptPtr<<<gridsize, blocksize, 0, stream[i]>>>(ctxgpu, &filegpu[offset], numBlocksPerStream);
		hipMemcpyAsync(&file[offset], &filegpu[offset], streamSize, hipMemcpyDeviceToHost, stream[i]);
	}

	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	hash_decrypted = hash(file, numblocks);

	hipEventElapsedTime(&cudaRuntime, start, stop);
	rate = (filesize*1e3)/(cudaRuntime);

	printf("Decryption done!\n");
	printf("Time taken: %lf milliseconds\n", cudaRuntime);
	printf("Average speed: %lf MB/s\n", rate/MEGABYTE);
	printf("Decrypted hash = %llx\n", (unsigned long long)hash_decrypted);

	// Check
	if (hash_decrypted == hash_original)
		printf("Hashes match! PASSED!\n");
	else
		printf("Hashes mismatch! FAILED!\n");

	//__________DONE__________
	blowfish_clean(context);
	hipHostFree(file);
	hipFree(filegpu);
	return 0;
}
